#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


using namespace std;

constexpr size_t ELEMENTS = 2048;

__global__ void vecadd(const int *A, const int *B, int *C)
{
	// Get block index
	unsigned int block_idx = blockIdx.x;
	// Get thread index
	unsigned int thread_idx = threadIdx.x;
	// Get the number of threads per block
	unsigned int block_dim = blockDim.x;
	// Get the thread's unique ID - (block_idx * block_dim) + thread_idx;
	unsigned int idx = (block_idx * block_dim) + thread_idx;
	// Add corresponding locations of A and B and store in C
	C[idx] = A[idx] + B[idx];
}

int main(int argc, char **argv)
{
	// Create host memory
	auto data_size = sizeof(int) * ELEMENTS;
	vector<int> A(ELEMENTS);    // Input aray
	vector<int> B(ELEMENTS);    // Input array
	vector<int> C(ELEMENTS);    // Output array

	// Initialise input data
	for (unsigned int i = 0; i < ELEMENTS; ++i)
		A[i] = B[i] = i;

	// Declare buffers
	int *buffer_A, *buffer_B, *buffer_C;

	// Initialise buffers
	hipMalloc((void**)&buffer_A, data_size);
	hipMalloc((void**)&buffer_B, data_size);
	hipMalloc((void**)&buffer_C, data_size);

	// Write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_B, &B[0], data_size, hipMemcpyHostToDevice);

	// Write host data to device
	hipMemcpy(buffer_A, &A[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_B, &B[0], data_size, hipMemcpyHostToDevice);

	// Run kernel with one thread for each element
	// First value is number of blocks, second is threads per block.  Max 1024 threads per block
	vecadd<<<ELEMENTS / 1024, 1024>>>(buffer_A, buffer_B, buffer_C);

	// Wait for kernel to complete
	hipDeviceSynchronize();

	// Read output buffer back to the host
	hipMemcpy(&C[0], buffer_C, data_size, hipMemcpyDeviceToHost);

	// Clean up resources
	hipFree(buffer_A);
	hipFree(buffer_B);
	hipFree(buffer_C);

	// Test that the results are correct
	for (int i = 0; i < 2048; ++i)
		if (C[i] != i + i)
			cout << "Error: " << i << endl;

	cout << "Finished" << endl;

	return 0;
}